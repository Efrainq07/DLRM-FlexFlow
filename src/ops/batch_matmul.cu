// Copyright 2020 Facebook
#include "model.h"
#include "cuda_helper.h"
#include <iostream>

Tensor FFModel::batch_matmul(std::string name,
                       const Tensor& input1, const Tensor& input2,
                       const bool trans1,
                       const bool trans2)
{
  BatchMatmul *bmm = new BatchMatmul(*this, name, input1, input2, trans1, trans2);
  layers.push_back(bmm);
  return bmm->output;
}


BatchMatmul::BatchMatmul(
    FFModel& model,
    const std::string& pcname,
    const Tensor& input1,
    const Tensor& input2,
    const bool trans1,
    const bool trans2
): Op(pcname, input1, input2){
    ArgumentMap argmap;
    // Retrive the task indexspace for the op
    task_is = model.get_or_create_task_is(pcname);
    Context ctx = model.config.lg_ctx;
    Runtime* runtime = model.config.lg_hlr;
    FieldSpace fs = model.config.field_space;


    // dimension in tensor constructor is ordered by `d,m,k`
    // but within the tensor object the dimensio is ordered by `k,m,d`
    // where the outmost dimension is d
    int d = input1.adim[2];
    int m = input1.adim[1];
    int n = input2.adim[1];
    int k = input1.adim[0];
    const int dims[] = {d,n,m};
    printf("batch_matmul inputs:\n");
    printf("input 1 shape d(%d) k(%d) m(%d)\n", d,k,m);
    printf("input 2 shape d(%d) k(%d) n(%d)\n", d,k,n);
    transpose_1_flag = trans1;
    transpose_2_flag = trans2;
    transpose_1 = trans1 ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    transpose_2 = trans2 ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    const int tensor_obj_n_dim = 3;
    // create 3-dimensional output tensor for this layer to hold the results
    output = model.create_tensor<tensor_obj_n_dim>(dims, "batch_matmul", DT_FLOAT);

    // Compute partition bound for input
    // TODO the input partition check can be refactored into a helper function
    Domain domain = runtime->get_index_space_domain(ctx, task_is);
    Rect<tensor_obj_n_dim> part_rect = domain;
    Rect<tensor_obj_n_dim> input1_rect = runtime->get_index_partition_color_space(
        ctx, input1.part.get_index_partition());
    if (input1_rect == part_rect) {
        input_lps[0] = input1.part;
        input_grad_lps[0] = input1.part_grad;
    } else {
        model.create_disjoint_partition<tensor_obj_n_dim>(
            input1,
            IndexSpaceT<3>(task_is),
            input_lps[0],
            input_grad_lps[0]
        );
    }
    Rect<tensor_obj_n_dim> input2_rect = runtime->get_index_partition_color_space(
        ctx, input2.part.get_index_partition());
    if (input2_rect == part_rect) {
        input_lps[1] = input2.part;
        input_grad_lps[1] = input2.part_grad;
    } else {
        model.create_disjoint_partition<tensor_obj_n_dim>(
            input2,
            IndexSpaceT<3>(task_is),
            input_lps[1],
            input_grad_lps[1]
        );
    }




    // move this one outside the constructor and initialize the output tensor outisde the constructor with a dummy initializer


    // initialize the output gradients here temporarily , we dont have to do this once we connect the layer to a loss layer
    // or receive the gradients from previous layer (in this case the gradients will be initialized/handled by previous layer)
    // current impl only supports 3 dimensional batch matmul , outter dimension is sample dimension
    Rect<3> rect = runtime->get_index_space_domain(ctx, task_is);
    int idx = 0;
    // seems like there are 2 ways to construct argument maps
    for (PointInRectIterator<3> it(rect); it(); it++) {
        OpMeta* mp = meta[idx++];
        argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
    }
    Domain output_grad_domain = runtime->get_index_partition_color_space(
        ctx, output.part_grad.get_index_partition());
    IndexSpace output_grad_task_is = model.get_or_create_task_is(output_grad_domain);
    // HACK: launch intialize gradients task, this one is used in weights gradients, we are not supposed to
    // initialize non-weights gradients in the layer (should receive it from parent layer)
    IndexLauncher launcher(ZERO_INIT_TASK_ID, output_grad_task_is,
                           TaskArgument(NULL, 0), argmap,
                           Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                           FFConfig::get_hash_id(std::string("init output gradients")));
    launcher.add_region_requirement(
        RegionRequirement(output.part_grad, 0/*projection*/,
                          WRITE_ONLY, EXCLUSIVE, output.region_grad));
    launcher.add_field(0, FID_DATA);
    runtime->execute_index_space(ctx, launcher);

}


void BatchMatmul::init(const FFModel& ff){
    ArgumentMap argmap;
    Context ctx = ff.config.lg_ctx;
    Runtime* runtime = ff.config.lg_hlr;
    // currently only support 3 dimensional batch matmul , outter dimension is sample dimension
    Rect<3> rect = runtime->get_index_space_domain(ctx, task_is);
    int idx = 0;
    for (PointInRectIterator<3> it(rect); it(); it++) {
        FFHandler handle = ff.handlers[idx++];
        argmap.set_point(*it, TaskArgument(&handle, sizeof(FFHandler)));
    }
    IndexLauncher launcher(BATCHMATMUL_INIT_TASK_ID, task_is,
        TaskArgument(this, sizeof(BatchMatmul)), argmap,
        Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
        FFConfig::get_hash_id(std::string(name)));
    launcher.add_region_requirement(
    RegionRequirement(output.part, 0/*projection id*/,
        WRITE_ONLY, EXCLUSIVE, output.region));
    launcher.add_field(0, FID_DATA);
    launcher.add_region_requirement(
    RegionRequirement(inputs[0].part, 0/*projection id*/,
        READ_ONLY, EXCLUSIVE, inputs[0].region));
    launcher.add_field(1, FID_DATA);
    launcher.add_region_requirement(
    RegionRequirement(inputs[1].part, 0/*projection id*/,
        READ_ONLY, EXCLUSIVE, inputs[1].region));
    launcher.add_field(2, FID_DATA);
    FutureMap fm = runtime->execute_index_space(ctx, launcher);
    fm.wait_all_results();
    idx = 0;
    for (PointInRectIterator<3> it(rect); it(); it++) {
        meta[idx++] = fm.get_result<OpMeta*>(*it);
    }


}


/*
  regions[0](O): output
  regions[1](I): input1
  regions[2](I): input2
*/
OpMeta* BatchMatmul::init_task(const Task *task,
    const std::vector<PhysicalRegion> &regions,
    Context ctx, Runtime *runtime)
{
    assert(regions.size() == 3);
    assert(task->regions.size() == 3);
    FFHandler handle = *((const FFHandler*) task->local_args);
    //TensorAccessorR<float, 2> acc_input(
    //    regions[0], task->regions[0], FID_DATA, ctx, runtime);
    TensorAccessorW<float, 3> acc_output(
        regions[0], task->regions[0], FID_DATA, ctx, runtime,
        false/*readOutput*/);
    TensorAccessorR<float, 3> input1(
        regions[1], task->regions[1], FID_DATA, ctx, runtime);
    TensorAccessorR<float, 3> input2(
        regions[2], task->regions[2], FID_DATA, ctx, runtime);


    /*
    input1 (k,m,d)
    input2 (k,n,d)
    output (n,m,d)
    */
    int k = input1.rect.hi[0] - input1.rect.lo[0] + 1;
    int m = acc_output.rect.hi[1] - acc_output.rect.lo[1] + 1;
    int n = acc_output.rect.hi[0] - acc_output.rect.lo[0] + 1;
    int batch_stride_a = input1.rect.hi[2] - input1.rect.lo[2] + 1;
    int batch_stride_b = input2.rect.hi[2] - input2.rect.lo[2] + 1;
    int batch_stride_c = acc_output.rect.hi[2] - acc_output.rect.lo[2] + 1;



    BatchMatmulMeta* bmm_meta = new BatchMatmulMeta(handle);
    printf("init batch_matmul (input): batdh_dim(%d) k(%d) m(%d) n(%d)\n", batch_stride_a, k, m, n);

    checkCUDNN(hipdnnCreateTensorDescriptor(&bmm_meta->outputTensor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(bmm_meta->outputTensor,
                        HIPDNN_TENSOR_NCHW,
                        HIPDNN_DATA_FLOAT,
                        batch_stride_a, 1, m, n));
    return bmm_meta;
}


void BatchMatmul::backward(const FFModel& ff){
    ArgumentMap argmap;
    Context ctx = ff.config.lg_ctx;
    Runtime* runtime = ff.config.lg_hlr;
    // currently only support 3 dimensional batch matmul , outter dimension is sample dimension
    Rect<3> rect = runtime->get_index_space_domain(ctx, task_is);
    int idx = 0;
    for (PointInRectIterator<3> it(rect); it(); it++) {
        OpMeta* mp = meta[idx++];
        argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
    }

    /*
    CHECK THIS LATERCHECK THIS LATERCHECK THIS LATERCHECK THIS LATERCHECK THIS LATERCHECK THIS LATER
    */
  IndexLauncher launcher(BATCHMATMUL_BWD_TASK_ID, task_is,
                         TaskArgument(this, sizeof(BatchMatmul)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
      RegionRequirement(output.part_grad, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, output.region_grad));
  launcher.add_field(0, FID_DATA);
    // input1 grad
    launcher.add_region_requirement(
                        RegionRequirement(input_grad_lps[0], 0/*projection id*/,
                        WRITE_ONLY, EXCLUSIVE, inputs[0].region_grad));
    launcher.add_field(1, FID_DATA);
    // input 2 grad
    launcher.add_region_requirement(
                        RegionRequirement(input_grad_lps[1], 0/*projection id*/,
                        WRITE_ONLY, EXCLUSIVE, inputs[1].region_grad));
    launcher.add_field(2, FID_DATA);
    // input1
    launcher.add_region_requirement(
                        RegionRequirement(inputs[0].part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
    launcher.add_field(3, FID_DATA);
    // input2
    launcher.add_region_requirement(
                        RegionRequirement(inputs[1].part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[1].region));
    launcher.add_field(4, FID_DATA);


    runtime->execute_index_space(ctx, launcher);
}



/*
  regions[0](O): output
  regions[1](I): input1
  regions[2](I): input2
*/
void BatchMatmul::forward_task(
    const Task *task,
    const std::vector<PhysicalRegion> &regions,
    Context ctx, Runtime *runtime
    )
{
    const BatchMatmul* bm = (BatchMatmul*) task->args;
    float alpha = 1.0f, beta = 0.0f;
    const BatchMatmulMeta* lm = *((BatchMatmulMeta**) task->local_args);
    const int batch_tensor_dim = 3;
    TensorAccessorW<float, batch_tensor_dim> acc_output(
        regions[0], task->regions[0], FID_DATA, ctx, runtime,
        false/*readOutput*/);
    TensorAccessorR<float, batch_tensor_dim> acc_input1(
        regions[1], task->regions[1], FID_DATA, ctx, runtime);

    TensorAccessorR<float, batch_tensor_dim> acc_input2(
        regions[2], task->regions[2], FID_DATA, ctx, runtime);


    int k = acc_input1.rect.hi[0] - acc_input1.rect.lo[0] + 1;
    int m = acc_output.rect.hi[1] - acc_output.rect.lo[1] + 1;
    int n = acc_output.rect.hi[0] - acc_output.rect.lo[0] + 1;
    int batch_stride_a = acc_input1.rect.hi[2] - acc_input1.rect.lo[2] + 1;
    int batch_stride_b = acc_input2.rect.hi[2] - acc_input2.rect.lo[2] + 1;
    int batch_stride_c = acc_output.rect.hi[2] - acc_output.rect.lo[2] + 1;
    printf("k:%d m:%d n:%d batch_stride_a:%d batch_stride_b:%d batch_stride_c:%d\n", k, m,n,batch_stride_a, batch_stride_b, batch_stride_c);
    printf("cuBLAS initializing...\n");
    #ifndef DISABLE_LEGION_CUDA_HIJACK
        hipStream_t stream;
        checkCUDA(hipStreamCreate(&stream));
        checkCUDA(hipblasSetStream(lm->handle.blas, stream));
        checkCUDNN(hipdnnSetStream(lm->handle.dnn, stream));
    #endif

    // because cublas is row major ordering, so leading dimension is the reduction dimension
    checkCUDA(
        hipblasSgemmStridedBatched(
            lm->handle.blas,
            bm->transpose_1,
            bm->transpose_2,
            m, n, k,
            &alpha,
            acc_input1.ptr, k,
            m*k,
            acc_input2.ptr, k,
            k*n,
            &beta,
            acc_output.ptr, m,
            m*n,
            batch_stride_a)
    );



    print_tensor<3, float>(acc_input1.ptr, acc_input1.rect, "[BatchMatmul:forward:input1]");
    print_tensor<3, float>(acc_input2.ptr, acc_input2.rect, "[BatchMatmul:forward:input2]");
    print_tensor<3, float>(acc_output.ptr, acc_output.rect, "[BatchMatmul:forward:output]");
}


void BatchMatmul::forward(const FFModel& ff){

    ArgumentMap argmap;
    Context ctx = ff.config.lg_ctx;
    Runtime* runtime = ff.config.lg_hlr;
    // currently only support 3 dimensional batch matmul , outter dimension is sample dimension
    Rect<3> rect = runtime->get_index_space_domain(ctx, task_is);
    int idx = 0;
    for (PointInRectIterator<3> it(rect); it(); it++) {
        OpMeta* mp = meta[idx++];
        argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
        // FFHandler handle = ff.handlers[idx++];
        // argmap.set_point(*it, TaskArgument(&handle, sizeof(FFHandler)));
    }

    IndexLauncher launcher(BATCHMATMUL_FWD_TASK_ID, task_is,
                           TaskArgument(this, sizeof(BatchMatmul)), argmap,
                           Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                           FFConfig::get_hash_id(std::string(name)));
    launcher.add_region_requirement(
        RegionRequirement(output.part, 0/*projection id*/,
                          WRITE_ONLY, EXCLUSIVE, output.region));
    launcher.add_field(0, FID_DATA);
    for (int i = 0; i < 2; i++) {
      launcher.add_region_requirement(
          RegionRequirement(input_lps[i], 0/*projection id*/,
            READ_ONLY, EXCLUSIVE, inputs[i].region));
      launcher.add_field(i+1, FID_DATA);
    }
    runtime->execute_index_space(ctx, launcher);
}







/*
  regions[0](O): output_grad
  regions[1](I): input1_grad
  regions[2](I): input2_grad
  regions[3](I): input1
  regions[4](I): input2
*/
void BatchMatmul::backward_task(
    const Task *task,
    const std::vector<PhysicalRegion> &regions,
    Context ctx, Runtime *runtime
    )
{
    const BatchMatmul* bm = (BatchMatmul*) task->args;
    float alpha = 1.0f, beta = 0.0f;
    const BatchMatmulMeta* lm = *((BatchMatmulMeta**) task->local_args);
    const int batch_tensor_dim = 3;
    TensorAccessorR<float, batch_tensor_dim> acc_output_grad(
        regions[0], task->regions[0], FID_DATA, ctx, runtime);
    TensorAccessorW<float, batch_tensor_dim> acc_input1_grad(
        regions[1], task->regions[1], FID_DATA, ctx, runtime, false/*readOutput*/);

    TensorAccessorW<float, batch_tensor_dim> acc_input2_grad(
        regions[2], task->regions[2], FID_DATA, ctx, runtime, false/*readOutput*/);
    TensorAccessorR<float, batch_tensor_dim> acc_input1(
        regions[3], task->regions[3], FID_DATA, ctx, runtime);

    TensorAccessorR<float, batch_tensor_dim> acc_input2(
        regions[4], task->regions[4], FID_DATA, ctx, runtime);


    int k = acc_input1_grad.rect.hi[0] - acc_input1_grad.rect.lo[0] + 1;
    int m = acc_output_grad.rect.hi[1] - acc_output_grad.rect.lo[1] + 1;
    int n = acc_output_grad.rect.hi[0] - acc_output_grad.rect.lo[0] + 1;
    int batch_stride_a = acc_input1_grad.rect.hi[2] - acc_input1_grad.rect.lo[2] + 1;
    int batch_stride_b = acc_input2_grad.rect.hi[2] - acc_input2_grad.rect.lo[2] + 1;
    int batch_stride_c = acc_output_grad.rect.hi[2] - acc_output_grad.rect.lo[2] + 1;
    printf("k:%d m:%d n:%d batch_stride_a:%d batch_stride_b:%d batch_stride_c:%d\n", k, m,n,batch_stride_a, batch_stride_b, batch_stride_c);

    #ifndef DISABLE_LEGION_CUDA_HIJACK
    hipStream_t stream;
    checkCUDA(hipStreamCreate(&stream));
    checkCUDA(hipblasSetStream(lm->handle.blas, stream));
    checkCUDNN(hipdnnSetStream(lm->handle.dnn, stream));
    #endif
    if (bm->transpose_1_flag) {
        if (bm->transpose_2_flag) {
            // A'B':
            // dA = B'G', dB = G'A'
            // checkCUDA(hipblasSgemmStridedBatched(lm->handle.blas,
            //                 HIPBLAS_OP_T, HIPBLAS_OP_T,
            //                 k,m,n,
            //                 &alpha,
            //                 acc_input2.ptr, k,
            //                 k*n,
            //                 acc_output_grad.ptr, m,
            //                 m*n,
            //                 &beta,
            //                 acc_input1_grad.ptr, k,
            //                 m*k,
            //                 batch_stride_a));
            // checkCUDA(hipblasSgemmStridedBatched(lm->handle.blas,
            //                 HIPBLAS_OP_T, HIPBLAS_OP_T,
            //                 n,k,m,
            //                 &alpha,
            //                 acc_output_grad.ptr, m,
            //                 m*n,
            //                 acc_input1.ptr, k,
            //                 m*k,
            //                 &beta,
            //                 acc_input2_grad.ptr, k,
            //                 k*n,
            //                 batch_stride_a));
            // not implemented
            throw 255;
        }
        else {
            // A'B:
            // dA = BG', dB = AG
            checkCUDA(hipblasSgemmStridedBatched(lm->handle.blas,
                                        HIPBLAS_OP_N, HIPBLAS_OP_T,
                                        k,m,n,
                                        &alpha,
                                        acc_input2.ptr, k,
                                        k*n,
                                        acc_output_grad.ptr, m,
                                        m*n,
                                        &beta,
                                        acc_input1_grad.ptr, k,
                                        m*k,
                                        batch_stride_a));
            checkCUDA(hipblasSgemmStridedBatched(lm->handle.blas,
                                        HIPBLAS_OP_N, HIPBLAS_OP_N,
                                        k,n,m,
                                        &alpha,
                                        acc_input1.ptr, k,
                                        m*k,
                                        acc_output_grad.ptr, m,
                                        m*n,
                                        &beta,
                                        acc_input2_grad.ptr, k,
                                        k*n,
                                        batch_stride_a));
        }
    } else {
        if (bm->transpose_2_flag) {
            // AB':
            // dA = GB, dB = G'A
            // not implemented
            throw 255;

        }
        else {
            // AB:
            // dA = GB', dB = A'G
            // not implemented
            throw 255;
        }
    }


    print_tensor<3, float>(acc_output_grad.ptr, acc_output_grad.rect, "[BatchMatmul:backward:acc_output_grad]");
    print_tensor<3, float>(acc_input1_grad.ptr, acc_input1_grad.rect, "[BatchMatmul:backward:input1_gard]");
    print_tensor<3, float>(acc_input1_grad.ptr, acc_input1_grad.rect, "[BatchMatmul:backward:input2_gard]");


}



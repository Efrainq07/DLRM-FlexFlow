// Copyright 2020 Facebook
#include "model.h"
#include "cuda_helper.h"
#include <iostream>

Tensor FFModel::transpose(const Tensor& input)
{
  Transpose *trans = new Transpose(*this, input);
  layers.push_back(trans);
  return trans->outputs[0];
}

Transpose::Transpose(FFModel& model,
                     const Tensor& _input)
: Op(model, OP_TRANSPOSE, "Transpose_", _input), profiling(model.config.profiling)
{
}

Tensor Transpose::init_inout(FFModel& model, const Tensor& _input)
{
  // TODO: This function is designed for support functional APIs
  // as used in PyTorch and Keras
  // TO BE IMPLEMENTED...
  assert(false);
  return Tensor();
}

void Transpose::create_weights(FFModel& model)
{
  // Do nothing
}

void Transpose::create_output_and_partition(FFModel& model)
{
  // Retrive the task indexspace for the op
  std::string pcname = name;
  task_is = IndexSpaceT<3>(model.get_or_create_task_is(3, pcname));
  {
    int k = inputs[0].adim[0];
    int m = inputs[0].adim[1];
    int d = inputs[0].adim[2];
    const int dims[] = {d,k,m};
    outputs[0] = model.create_tensor<3>(dims, (IndexSpaceT<3>)task_is, DT_FLOAT);
    outputs[0].owner_op = this;
    outputs[0].owner_idx = 0;
  }

  model.create_data_parallel_partition_with_diff_dims<3, 3>(
    inputs[0], IndexSpaceT<3>(task_is), input_lps[0], input_grad_lps[0]);
}

void Transpose::init(const FFModel& ff) {
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  // currently only support 3 dimensional transpose , outter dimension is sample dimension
  Rect<3> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<3> it(rect); it(); it++) {
    FFHandler handle = ff.handlers[idx++];
    argmap.set_point(*it, TaskArgument(&handle, sizeof(FFHandler)));
  }
  IndexLauncher launcher(TRANSPOSE_INIT_TASK_ID, task_is,
    TaskArgument(this, sizeof(Transpose)), argmap,
    Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
    FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
  RegionRequirement(outputs[0].part, 0/*projection id*/,
    WRITE_ONLY, EXCLUSIVE, outputs[0].region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
  RegionRequirement(inputs[0].part, 0/*projection id*/,
    READ_WRITE, EXCLUSIVE, inputs[0].region));
  launcher.add_field(1, FID_DATA);
  
  FutureMap fm = runtime->execute_index_space(ctx, launcher);
  fm.wait_all_results();
  idx = 0;
  for (PointInRectIterator<3> it(rect); it(); it++) {
    meta[idx++] = fm.get_result<OpMeta*>(*it);
  }
}

OpMeta* Transpose::init_task(const Task *task,
  const std::vector<PhysicalRegion> &regions,
  Context ctx, Runtime *runtime)
{
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  const Transpose* bm = (Transpose*) task->args;
  FFHandler handle = *((const FFHandler*) task->local_args);
  TensorAccessorW<float, 3> acc_output(
    regions[0], task->regions[0], FID_DATA, ctx, runtime,
    false/*readOutput*/);
  TensorAccessorR<float, 3> input1(
    regions[1], task->regions[1], FID_DATA, ctx, runtime);
  
  /*
  input1 (k,m,d)
  output (m,k,d)
  */
  int k = input1.rect.hi[0] - input1.rect.lo[0] + 1;
  int m = input1.rect.hi[1] - input1.rect.lo[1] + 1;
  int batch_stride_a = input1.rect.hi[2] - input1.rect.lo[2] + 1;
  int batch_stride_c = acc_output.rect.hi[2] - acc_output.rect.lo[2] + 1;
  TransposeMeta* bmm_meta = new TransposeMeta(handle);
  if (bm->profiling){ 
    printf("init transpose (input): batdh_dim(%d) k(%d) m(%d) \n", batch_stride_a, k, m);
  }
  return bmm_meta;
}

void Transpose::forward(const FFModel& ff) {
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  // currently only support 3 dimensional transpose , outter dimension is sample dimension
  Rect<3> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<3> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }
  IndexLauncher launcher(TRANSPOSE_FWD_TASK_ID, task_is,
    TaskArgument(this, sizeof(Transpose)), argmap,
    Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
    FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
    RegionRequirement(inputs[0].part, 0/*projection id*/,
      READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
    RegionRequirement(outputs[0].part, 0/*projection id*/,
      WRITE_ONLY, EXCLUSIVE, outputs[0].region));
  launcher.add_field(1, FID_DATA);
  runtime->execute_index_space(ctx, launcher);
}


/*
  regions[0](I): input
  regions[1](O): output
*/
void Transpose::forward_task(const Task *task,
                         const std::vector<PhysicalRegion> &regions,
                         Context ctx, Runtime *runtime) {
  const Transpose* bm = (Transpose*) task->args;
  float alpha = 1.0f, beta = 0.0f;
  const TransposeMeta* lm = *((TransposeMeta**) task->local_args);
  const int batch_tensor_dim = 3;
  TensorAccessorR<float, batch_tensor_dim> acc_input(
    regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, batch_tensor_dim> acc_output(
    regions[1], task->regions[1], FID_DATA, ctx, runtime,
    false/*readOutput*/);
  /*
  shape d,m,k
  order d(2),m(1),k(0)
  axis    k,m,d
  index   2 1 0
  input1 (d,m,k)
  output (d,k,m)
  */
  int k = acc_input.rect.hi[0] - acc_input.rect.lo[0] + 1;
  int m = acc_input.rect.hi[1] - acc_input.rect.lo[1] + 1;
  int batch_stride_a = acc_input.rect.hi[2] - acc_input.rect.lo[2] + 1;
  int batch_stride_b = acc_output.rect.hi[2] - acc_output.rect.lo[2] + 1;
  if (bm->profiling){ 
    printf("k:%d m:%d batch_stride_input:%d batch_stride_output:%d\n", k, m, batch_stride_a, batch_stride_b);
    printf("cuBLAS initializing...\n");
  }
#ifndef DISABLE_LEGION_CUDA_HIJACK
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDA(hipblasSetStream(lm->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(lm->handle.dnn, stream));
#endif
  for(int batch_count = 0; batch_count < batch_stride_a; batch_count++) {
    int batch_stride = m*k;
    int offset = batch_count * batch_stride;
    checkCUDA(
      hipblasSgeam(
        lm->handle.blas,
        HIPBLAS_OP_T,
        HIPBLAS_OP_N, /*although we are not using this but still have to pass in correct shape*/
        m,k,
        &alpha,
        acc_input.ptr+offset, k,
        &beta,
        acc_input.ptr+offset, m, /*although we are not using this but still have to pass in correct shape*/
        acc_output.ptr+offset, m
      )
    );
  }
  if (bm->profiling){ 
    printf("input1 d:%d k:%d m:%d\n", batch_stride_a, k, m );
    print_tensor<3, float>(acc_input.ptr, acc_input.rect, "[Transpose:forward:input]");
    print_tensor<3, float>(acc_output.ptr, acc_output.rect, "[Transpose:forward:output]");
  }
}

void Transpose::backward(const FFModel& ff) {
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  // currently only support 3 dimensional transpose , outter dimension is sample dimension
  Rect<3> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<3> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }
  IndexLauncher launcher(TRANSPOSE_BWD_TASK_ID, task_is,
    TaskArgument(this, sizeof(Transpose)), argmap,
    Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
    FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
    RegionRequirement(inputs[0].part_grad, 0/*projection id*/,
      WRITE_ONLY, EXCLUSIVE, inputs[0].region_grad));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
    RegionRequirement(outputs[0].part_grad, 0/*projection id*/,
      READ_ONLY, EXCLUSIVE, outputs[0].region_grad));
  launcher.add_field(1, FID_DATA);
  runtime->execute_index_space(ctx, launcher);
}

void Transpose::backward_task(
                        const Task *task,
                        const std::vector<PhysicalRegion> &regions,
                        Context ctx, Runtime *runtime
                        ){
  const Transpose* bm = (Transpose*) task->args;
  float alpha = 1.0f, beta = 0.0f;
  const TransposeMeta* lm = *((TransposeMeta**) task->local_args);
  const int batch_tensor_dim = 3;
  TensorAccessorW<float, batch_tensor_dim> acc_input(
    regions[0], task->regions[0], FID_DATA, ctx, runtime,
    false/*readOutput*/);
  TensorAccessorR<float, batch_tensor_dim> acc_output(
    regions[1], task->regions[1], FID_DATA, ctx, runtime);
  /*
  shape d,m,k
  order d(2),m(1),k(0)
  axis    k,m,d
  index   2 1 0
  input1 (d,m,k)
  output (d,k,m)
  */
  int k = acc_input.rect.hi[0] - acc_input.rect.lo[0] + 1;
  int m = acc_input.rect.hi[1] - acc_input.rect.lo[1] + 1;
  int batch_stride_a = acc_input.rect.hi[2] - acc_input.rect.lo[2] + 1;
  int batch_stride_b = acc_output.rect.hi[2] - acc_output.rect.lo[2] + 1;
  if (bm->profiling){ 
    printf("k:%d m:%d batch_stride_input:%d batch_stride_output:%d\n", k, m, batch_stride_a, batch_stride_b);
    printf("cuBLAS initializing...\n");
  }
#ifndef DISABLE_LEGION_CUDA_HIJACK
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDA(hipblasSetStream(lm->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(lm->handle.dnn, stream));
#endif
  for(int batch_count = 0; batch_count < batch_stride_a; batch_count++) {
    int batch_stride = m*k;
    int offset = batch_count * batch_stride;
    checkCUDA(
      hipblasSgeam(
        lm->handle.blas,
        HIPBLAS_OP_T,
        HIPBLAS_OP_N,
        k,m,
        &alpha,
        acc_output.ptr+offset, m,
        &beta,
        acc_output.ptr+offset, k,
        acc_input.ptr+offset, k
      )
    );
  }
  if (bm->profiling){ 
    printf("input1 d:%d k:%d m:%d\n", batch_stride_a, k, m );
    print_tensor<3, float>(acc_input.ptr, acc_input.rect, "[Transpose:backward:input]");
    print_tensor<3, float>(acc_output.ptr, acc_output.rect, "[Transpose:backward:output]");
  }
}

void Transpose::print_layer(const FFModel& model)
{}

bool Transpose::measure_compute_time(Simulator* sim,
                                     const ParallelConfig& pc,
                                     float& forward_time,
                                     float& backward_time)
{
  return false;
}


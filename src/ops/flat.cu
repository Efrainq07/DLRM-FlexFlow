/* Copyright 2018 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "model.h"
#include "cuda_helper.h"

Tensor FFModel::flat(std::string name, Tensor input)
{
  assert(input.numDim == 3);
  //assert(strategies.find(name) != strategies.end());
  //ParallelConfig pc = strategies[name];
  Flat *flat = new Flat(*this, name, input);
  layers.push_back(flat);
  return flat->output;
}

Flat::Flat(FFModel& model,
  const std::string& pcname,
  const Tensor& _input)
: Op(pcname, _input)
{
  task_is = IndexSpaceT<2>(model.get_or_create_task_is(2, pcname));
  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Rect<2> part_rect = runtime->get_index_space_domain(ctx, task_is);
  int num_par_c = part_rect.hi[0] - part_rect.lo[0] + 1;
  int num_par_n = part_rect.hi[1] - part_rect.lo[1] + 1;
  // Assert data parallelism for operators with dim changes
  assert(num_par_c == 1);
 
  int out_dim = _input.adim[0] * _input.adim[1] * _input.adim[2];
  int batch_size = _input.adim[3];
  // Create output tensor
  {
    const int dims[2] = {batch_size, out_dim};
    output = model.create_tensor<2>(dims, task_is, DT_FLOAT);
  }
  model.create_data_parallel_partition_with_diff_dims<3, 2>(
      _input, task_is, input_lps[0], input_grad_lps[0]);
}

OpMeta* Flat::init_task(const Task *task,
                        const std::vector<PhysicalRegion> &regions,
                        Context ctx, Runtime *runtime)
{
  FFHandler handler = *((const FFHandler*) task->local_args);
  FlatMeta* m = new FlatMeta(handler);
  return m;
}

void Flat::init(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<2> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<2> it(rect); it(); it++) {
    FFHandler handle = ff.handlers[idx++];
    argmap.set_point(*it, TaskArgument(&handle, sizeof(FFHandler)));
  }
  IndexLauncher launcher(FLAT_INIT_TASK_ID, task_is,
    TaskArgument(this, sizeof(Flat)), argmap,
    Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
    FFConfig::get_hash_id(std::string(name)));
  FutureMap fm = runtime->execute_index_space(ctx, launcher);
  fm.wait_all_results();
  idx = 0;
  for (PointInRectIterator<2> it(rect); it(); it++) {
    meta[idx++] = fm.get_result<OpMeta*>(*it);
  }
}

/*
  regions[0](I): input
  regions[1](O): output
*/  
void Flat::forward_task(const Task *task,
                        const std::vector<PhysicalRegion> &regions,
                        Context ctx, Runtime *runtime)
{
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  TensorAccessorR<float, 3> acc_input(
    regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 2> acc_output(
    regions[1], task->regions[1], FID_DATA, ctx, runtime,
    false/*readOutput*/);
  assert(acc_input.rect.volume() == acc_output.rect.volume());
  checkCUDA(hipMemcpyAsync(acc_output.ptr, acc_input.ptr,
    acc_input.rect.volume() * sizeof(float),
    hipMemcpyDeviceToDevice));
}

void Flat::forward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<2> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<2> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }
  IndexLauncher launcher(FLAT_FWD_TASK_ID, task_is,
    TaskArgument(NULL, 0), argmap,
    Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
    FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
    RegionRequirement(input_lps[0], 0/*projection id*/,
      READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
    RegionRequirement(output.part, 0/*projection id*/,
      WRITE_ONLY, EXCLUSIVE, output.region));
  launcher.add_field(1, FID_DATA);
  runtime->execute_index_space(ctx, launcher);
}

/*
  regions[0](O) : input_grad
  regions[1](I) : output_grad
*/
void Flat::backward_task(const Task *task,
                         const std::vector<PhysicalRegion> &regions,
                         Context ctx, Runtime *runtime)
{
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  TensorAccessorW<float, 4> acc_input_grad(
    regions[0], task->regions[0], FID_DATA, ctx, runtime,
    true/*readOutput*/);
  TensorAccessorR<float, 2> acc_output_grad(
    regions[1], task->regions[1], FID_DATA, ctx, runtime);
  assert(acc_input_grad.rect.volume() == acc_output_grad.rect.volume());
  checkCUDA(hipMemcpyAsync(acc_input_grad.ptr, acc_output_grad.ptr,
    acc_input_grad.rect.volume() * sizeof(float),
    hipMemcpyDeviceToDevice));
}

void Flat::backward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<2> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<2> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }
  IndexLauncher launcher(FLAT_BWD_TASK_ID, task_is,
    TaskArgument(NULL, 0), argmap,
    Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
    FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
    RegionRequirement(input_grad_lps[0], 0/*projection id*/,
      WRITE_ONLY, EXCLUSIVE, inputs[0].region_grad));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
    RegionRequirement(output.part_grad, 0/*projection id*/,
      READ_ONLY, EXCLUSIVE, output.region_grad));
  launcher.add_field(1, FID_DATA);
  runtime->execute_index_space(ctx, launcher);
}

